/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 To compile: 
    nvcc -arch=sm_60 reduction_kernel.cu
 To run with the array size 2^20, expo dist mean 5, and init seed 17:
    ./a.out 20 5 17
 */
#include <stdio.h>
#include <stdlib.h>

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<class T>
T reduceCPU(T *data, int size)
{
    T sum = data[0];
    T c = (T)0.0;

    for (int i = 1; i < size; i++)
    {
        T y = data[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }

    return sum;
}

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
__global__ void
qroot(double *g_idata, double *g_odata)
{
	 unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	 g_odata[i]= sqrt(sqrt(g_idata[i]));
} 
__global__ void
reduce(double *g_idata, double *g_odata, unsigned int n)
{
    double *sdata = SharedMemory<double>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


// CUDA Runtime
#include <hip/hip_runtime.h>

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

extern "C" double reduction(int n, int mean, int seed)
{
        int size = 1<<n;    // number of elements to reduce
        int maxThreads = 256;  // number of threads per block

        // create random input data on CPU
        unsigned int bytes = size * sizeof(double);

        double *h_idata = (double *) malloc(bytes);

        srand48(seed);
        for (int i=0; i<size; i++)
        {
                // h_idata[i] = 1.0; // for testing
                // expo dist with mean 5.0
                h_idata[i] = -mean * log(drand48());
        }

        int numBlocks = size / maxThreads;
        int numThreads = size;

        int smemSize = maxThreads * sizeof(double);

        // allocate mem for the result on host side
        double *h_odata = (double *) malloc(numBlocks*sizeof(double));

        // allocate device memory and data
        double  *d_idata = NULL;
        double *d_odata = NULL;

        checkCudaErrors(hipMalloc((void **) &d_idata, bytes));
        checkCudaErrors(hipMalloc((void **) &d_odata, numBlocks*sizeof(double)));

        // copy data directly to device memory
        checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
		qroot<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_idata);
        reduce<<<numBlocks,maxThreads,smemSize>>>(d_idata, d_odata, numThreads);

        int s=numBlocks;

        while (s > 1) {
            reduce<<<(s+maxThreads-1)/maxThreads,maxThreads,smemSize>>>(d_odata, d_odata, s);
            s = (s+maxThreads-1)/maxThreads;
        }

        checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(double), hipMemcpyDeviceToHost));

        printf("GPU sum : %f\n\n", h_odata[0]);
		double result = h_odata[0];

        checkCudaErrors(hipFree(d_idata));
        checkCudaErrors(hipFree(d_odata));

        double cpu_result = reduceCPU<double>(h_idata, size);
   
        printf("CPU sum : %f\n", cpu_result);
        
    return result;
}
